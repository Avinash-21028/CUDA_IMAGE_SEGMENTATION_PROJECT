#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <opencv2/core/cuda.hpp>

void colorizeImage(const cv::Mat& gray, cv::Mat& colorized) {
    #ifdef HAVE_OPENCV_CUDA
    cv::cuda::GpuMat d_gray(gray);
    cv::cuda::GpuMat d_colorized;
    cv::cuda::applyColorMap(d_gray, d_colorized, cv::COLORMAP_JET);
    d_colorized.download(colorized);
    #else
    cv::applyColorMap(gray, colorized, cv::COLORMAP_JET);
    #endif
}

void edgeDetection(const cv::Mat& gray, cv::Mat& edges) {
    #ifdef HAVE_OPENCV_CUDA
    cv::cuda::GpuMat d_gray(gray);
    cv::cuda::GpuMat d_edges;
    cv::cuda::Canny(d_gray, d_edges, 100, 200);
    d_edges.download(edges);
    #else
    cv::Canny(gray, edges, 100, 200);
    #endif
}

void sobelFilter(const cv::Mat& gray, cv::Mat& vertical, cv::Mat& horizontal) {
    #ifdef HAVE_OPENCV_CUDA
    cv::cuda::GpuMat d_gray(gray);
    cv::cuda::GpuMat d_vertical, d_horizontal;
    cv::cuda::Sobel(d_gray, d_vertical, CV_64F, 1, 0, 5);
    cv::cuda::Sobel(d_gray, d_horizontal, CV_64F, 0, 1, 5);
    d_vertical.download(vertical);
    d_horizontal.download(horizontal);
    #else
    cv::Sobel(gray, vertical, CV_64F, 1, 0, 5);
    cv::Sobel(gray, horizontal, CV_64F, 0, 1, 5);
    #endif
}
